#include "hip/hip_runtime.h"
// C standard libraries
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <stdlib.h>
#include <stdint.h>
#include <time.h>
#include <signal.h>

// Unix library
#include <unistd.h>
#include <pthread.h>

// Imported Libraries
#include "vips/vips.h"
#include <curl/curl.h>
#include <sw/redis++/redis++.h>
#include <pqxx/pqxx>

// C++ standard libraries
#include <iostream>
#include <string>
#include <memory>
#include <utility>
#include <iomanip>
#include <string>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

using namespace std;
using namespace pqxx;
using namespace sw::redis;

#define MAX_POSTGRES_QUERY_SIZE 1024*10
#define REDIS_QUEUE_NAME "data_image_qda:queue"
#define REDIS_WORKING_NAME "data_image_qda:working"
#define REDIS_COMPLETE_BASE_NAME "data_image_qda:complete"
#define REDIS_QUEUE_DETAILS_BASE_NAME "data_image_qda:queue_details"
#define REDIS_DETAILS_BASE_NAME "data_image_qda:details"
#define PNG_POOL_SIZE 32
#define PNG_UPLOAD_WAIT_TIME 30000

enum QualityCalcMethod { QualityLogExpSum = 0, QualityFirst = 1 };

/**
* Point-In-Polygons for a block using CUDA
*/
__global__ void PointsInPolygonsCUDA(int32_t start_lat, int32_t start_lng, int32_t image_size,
	int32_t num_polys, double quality_scale, enum QualityCalcMethod quality_calc_method, double quality_calc_value,
	uint8_t *image_mem, uint8_t *found_mem, uint8_t *all_blank, int32_t *vectors, double *poly_values, int32_t *vector_lengths) {

	int32_t x, y;
	x = blockIdx.x * blockDim.x + threadIdx.x;
	y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= image_size || y >= image_size) return;
	int32_t lat, lng, image_pos;
	lat = start_lat + (image_size-y-1);
	lng = start_lng + x;
	image_pos = y * image_size + x;

	double value = 0;
	for (int32_t i = 0, vector_ind = 0, j, intersections; i < num_polys; i++) {
		// Point in Polygon calculation
		for (intersections = 0, j = 0; j < vector_lengths[i]; j++, vector_ind += 4) {
			if (((vectors[vector_ind + 1]>lat) != (vectors[vector_ind + 3]>lat)) &&
				(lng < ((int64_t)vectors[vector_ind + 2] - vectors[vector_ind]) * ((int64_t)lat - vectors[vector_ind + 1]) / ((int64_t)vectors[vector_ind + 3] - vectors[vector_ind + 1]) + vectors[vector_ind])) {
				intersections++;
			}
		}
		if ((intersections & 1) == 1) {
			// Add power of value to sum
			if (quality_calc_method == QualityLogExpSum) {
				value += pow(quality_calc_value, poly_values[i]);
			}
			// Halt on first value
			else if (quality_calc_method == QualityFirst)
			{
				value = poly_values[i];
				break;
			}
		}
	}
	//printf("value: %f\n", value);
	
	if (value) {
		*all_blank = 0;
		switch (quality_calc_method) {
		case QualityLogExpSum:
			value = log(value) / log(quality_calc_value);
			break;
		case QualityFirst:
			found_mem[image_pos] = 1;
			break;
		default:
			break;
		}
	}
	value *= quality_scale;
	uint32_t fixed_value;
	if (value > UINT32_MAX)
		fixed_value = UINT32_MAX;
	else if (value < 0)
		fixed_value = 0;
	else
		fixed_value = value;

		
	image_pos = image_pos * 4;
	image_mem[image_pos] = (fixed_value >> 24) & 0xFF; // red
	image_mem[image_pos + 1] = (fixed_value >> 16) & 0xFF; // green
	image_mem[image_pos + 2] = (fixed_value >> 8) & 0xFF; // blue
	image_mem[image_pos + 3] = fixed_value & 0xFF; // alpha
}
//
/**
* Return an array of char poly_values that is equal to whether this point is in polygon[i]
*/

static int MemArrayToPngPointerWithFilter(VipsObject *scope, uint8_t *image_mem, uint8_t* found_mem, int32_t size, void **pngPointer, size_t *image_size) {
	VipsImage **ims = (VipsImage **) vips_object_local_array( scope, 7 );
  if(!(ims[0] = vips_image_new_from_memory( image_mem, 4 * size * size, size, size, 4, VIPS_FORMAT_UCHAR)))
    return -1;
  if(vips_copy(ims[0], ims+1, "bands", 1, "format", VIPS_FORMAT_UINT, NULL))
    return -1;
  if(found_mem) {
		// Apply median rank filtering on holes
    if(!(ims[2] = vips_image_new_from_memory( found_mem, size * size, size, size, 1, VIPS_FORMAT_UCHAR )) ||
      vips_median( ims[1], ims+3, 3, NULL ) ||
      vips_equal_const1( ims[2], ims+4, 1, NULL ) ||
      vips_ifthenelse( ims[4], ims[1], ims[3], ims+4, NULL ) )
        return -1;
  }
  else {
    ims[4] = ims[1];
    ims[1] = NULL;
  }
	if(vips_copy(ims[4], ims+5, "bands", 4, "format", VIPS_FORMAT_UCHAR, NULL))
    return -1;

  if( vips_pngsave_buffer(ims[5], pngPointer, image_size, "compression", 9, NULL) )
    return -1;

  return 0;
}

int PointInPolygonsImage(void **png_pointer, size_t *png_size, int32_t start_lat,
	int32_t start_lng, int32_t image_size, double quality_scale, enum QualityCalcMethod quality_calc_method,
	double quality_calc_value, int32_t *vectors, int32_t total_length, double *poly_values, int32_t num_polys, int32_t *vector_lengths) {
	hipStream_t stream;

	// Allocate device memory
	int32_t vectors_mem_size = total_length * 4 * sizeof(*vectors);
	int32_t vector_lengths_mem_size = num_polys * sizeof(*vector_lengths);
	int32_t values_mem_size = num_polys * sizeof(*poly_values);
	int32_t found_mem_size = image_size * image_size * sizeof(uint8_t);
	int32_t image_mem_size = found_mem_size * 4;
	int32_t *d_vectors;
	int32_t *d_vector_lengths;
	uint8_t *d_image_mem, *h_image_mem, *d_found_mem,
			*h_found_mem = NULL, *d_all_blank, h_all_blank[1] = { 1 };
	double *d_poly_values;
	const char *CUDA_DEVICE_ENV = getenv("CUDA_DEVICE_ENV");

	int32_t max_vector_lengths = 0;
	for (int32_t i = 0; i < num_polys; i++)
		if (vector_lengths[i] > max_vector_lengths)
			max_vector_lengths = vector_lengths[i];


	switch (quality_calc_method) {
	case QualityFirst:
		if (!(h_found_mem = reinterpret_cast<uint8_t *>(malloc(found_mem_size)))) {
			cerr << "Failed to allocate host found mem!" << endl;
			exit(EXIT_FAILURE);
		}
		checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_found_mem), found_mem_size));
		break;
	default:
		break;
	}


	if (!(h_image_mem = reinterpret_cast<uint8_t *>(malloc(image_mem_size)))) {
		cerr << "Failed to allocate host image mem!" << endl;
		exit(EXIT_FAILURE);
	}

	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_vectors), vectors_mem_size));
	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_vector_lengths), vector_lengths_mem_size));
	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_poly_values), values_mem_size));
	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_image_mem), image_mem_size));
	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_all_blank), sizeof(uint8_t)));
	// Allocate CUDA events that we'll use for timing
	// hipEvent_t start, stop;
	// checkCudaErrors(hipEventCreate(&start));
	// checkCudaErrors(hipEventCreate(&stop));

	checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

	// copy host memory to device
	checkCudaErrors(hipMemcpyAsync(d_vectors, vectors, vectors_mem_size, hipMemcpyHostToDevice, stream));
	checkCudaErrors(hipMemcpyAsync(d_poly_values, poly_values, values_mem_size, hipMemcpyHostToDevice, stream));
	checkCudaErrors(hipMemcpyAsync(d_vector_lengths, vector_lengths, vector_lengths_mem_size, hipMemcpyHostToDevice, stream));
	checkCudaErrors(hipMemcpyAsync(d_all_blank, h_all_blank, sizeof(uint8_t), hipMemcpyHostToDevice, stream));

	// Setup execution parameters
	dim3 threads(28, 28);
	if(CUDA_DEVICE_ENV && strcmp(CUDA_DEVICE_ENV, "production")) {
		// 2560 cores O_o
		threads.x = 64;
		threads.y = 40;
	}
	dim3 grid((image_size + threads.x - 1) / threads.x, (image_size + threads.y - 1) / threads.y);

	// Create and start timer
	// printf("Computing result using CUDA Kernel...\n");

	// Record the start event
	// checkCudaErrors(hipEventRecord(start, stream));
	// printf("Sending grid: [%d,%d]\n", grid.x, grid.y);
	// printf("Processing %d vectors %d times\n", total_length, image_size*image_size);
	// Performs warmup operation using matrixMul CUDA kernel
	PointsInPolygonsCUDA << < grid, threads, 0, stream >> > (start_lat, start_lng, image_size,
		num_polys, quality_scale, quality_calc_method, quality_calc_value,
		d_image_mem, d_found_mem, d_all_blank, d_vectors, d_poly_values, d_vector_lengths);
	hipError_t error = (hipGetLastError());

	if (error != hipSuccess)
	{
		fprintf(stderr, "Failed to launch kernel (error code '%s')!\n", hipGetErrorString(error));
	}

	// Record the stop event
	// checkCudaErrors(hipEventRecord(stop, stream));

	// Wait for the stop event to complete
	// checkCudaErrors(hipEventSynchronize(stop));

	// float msecTotal = 0.0f;
	// checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

	// Compute and print the performance
	// printf(
	// 	"Time= %.3f msec, GInt64OPS=%.3f\n",
	// 	msecTotal, (float)total_length*image_size*image_size/(msecTotal/1000)/1024/1024/1024);

	// Copy result from device to host
	checkCudaErrors(hipMemcpyAsync(h_all_blank, d_all_blank, sizeof(char), hipMemcpyDeviceToHost, stream));
	checkCudaErrors(hipMemcpyAsync(h_image_mem, d_image_mem, image_mem_size, hipMemcpyDeviceToHost, stream));
	switch (quality_calc_method) {
	case QualityFirst:
		checkCudaErrors(hipMemcpyAsync(h_found_mem, d_found_mem, found_mem_size, hipMemcpyDeviceToHost, stream));
		checkCudaErrors(hipFree(d_found_mem));
		break;
	default:
		break;
	}
	checkCudaErrors(hipStreamSynchronize(stream));

	// Clean up memory
	checkCudaErrors(hipFree(d_all_blank));
	checkCudaErrors(hipFree(d_vectors));
	checkCudaErrors(hipFree(d_image_mem));
	checkCudaErrors(hipFree(d_poly_values));
	checkCudaErrors(hipFree(d_vector_lengths));
	// checkCudaErrors(hipEventDestroy(start));
	// checkCudaErrors(hipEventDestroy(stop));

	if(*h_all_blank) {
		*png_pointer = NULL;
		free(h_image_mem);
		switch (quality_calc_method) {
		case QualityFirst:
			free(h_found_mem);
			break;
		default:
			break;
		}
		return EXIT_SUCCESS;
	}

	VipsObject *scope;
	scope = VIPS_OBJECT( vips_image_new() );
	if(MemArrayToPngPointerWithFilter(scope, h_image_mem, h_found_mem, image_size, png_pointer, png_size))
		vips_error_exit( NULL );
  g_object_unref( scope );

	free(h_image_mem);
	switch (quality_calc_method) {
	case QualityFirst:
		free(h_found_mem);
		break;
	default:
		break;
	}
	//if (correct) {
	return EXIT_SUCCESS;
	//} else {
	//    return EXIT_FAILURE;
	//}
}

void JsonGeometryToVectors(string json_array, double multiply_const, int32_t **vectors, int32_t *total_length, int32_t *vector_length) {
	size_t pos = 1, coord_ind, end_pos, current_pos;
	*vector_length = 0;
	float lat_flt, lng_flt;
	int32_t lat_1, lng_1, lat_2, lng_2;
	while(json_array.find_first_of('[', pos) != string::npos) { // Polygon
		pos++;
		while(json_array[pos] == '[') { // Polygon or Hole
			pos++;
			coord_ind = 0;
			while(json_array[pos] == '[') { // Coord
				pos++;
				end_pos = json_array.find_first_of(']', pos);
				sscanf(json_array.substr(pos, end_pos-pos+1).c_str(), "%f,%f]", &lng_flt, &lat_flt);
				lat_2 = multiply_const * lat_flt;
				lng_2 = multiply_const * lng_flt;
				if(coord_ind != 0) {
					(*total_length)++;
					(*vector_length)++;
					if(*total_length == 1) {
						*vectors = reinterpret_cast<int32_t *>(malloc(sizeof(**vectors) * 4));
					}
					else {
						*vectors = reinterpret_cast<int32_t *>(realloc(*vectors, sizeof(**vectors)*(*total_length)*4));
					}
					current_pos=(*total_length-1)*4;
					(*vectors)[current_pos] = lng_1;
					(*vectors)[current_pos+1] = lat_1;
					(*vectors)[current_pos+2] = lng_2;
					(*vectors)[current_pos+3] = lat_2;
				}
				lat_1 = lat_2;
				lng_1 = lng_2;
				pos = end_pos+2;
				coord_ind++;
			}
			pos++;
		}
		//Example (Pretend no whitespace)
		//[  Entire Geometry
		//	[ A Polygon
		//		[ A polygon or hole
		//			[-104.862892,39.745213],[-104.86039,39.742725],[-104.859138,39.742374],[-104.857269,39.740959],[-104.856567,39.739964],[-104.857697,39.73896],[-104.857925,39.737965],[-104.86097,39.735962],[-104.863129,39.733803],[-104.864136,39.734241],[-104.865135,39.73394],[-104.867683,39.735962],[-104.86821,39.736885],[-104.870064,39.737965],[-104.870186,39.738903],[-104.871124,39.739964],[-104.866142,39.744408],[-104.863136,39.745396],[-104.862892,39.745213]
		//		]
		//	]
		//]
		pos++;
	}
}

int RetrieveValuesFromPG(connection *C, string select_request, double multiply_const,
		int32_t **vectors, int32_t *total_length, double **poly_values, int32_t *num_polys, int32_t **vector_lengths) {
	try {
		/* Create a non-transactional object. */
		nontransaction N(*C);
		
		/* Execute SQL query */
		result R( N.exec( select_request ));
		*num_polys = R.size();
		(*poly_values) = reinterpret_cast<double *>(malloc(sizeof(**poly_values)*(*num_polys)));
		(*vector_lengths) = reinterpret_cast<int32_t *>(malloc(sizeof(**vector_lengths)*(*num_polys)));
		*total_length = 0;
		int32_t i = 0;
		for (result::const_iterator c = R.begin(); c != R.end(); ++c, i++) {
			(*poly_values)[i] = c[1].as<double>();
			JsonGeometryToVectors(c[0].as<string>(), multiply_const, vectors, total_length, (*vector_lengths)+i);
		}
	} catch (const exception &e) {
		cerr << e.what() << endl;
		return 1;
	}
	return 0;
}

struct WriteThis {
  char *readptr;
  size_t sizeleft;
};
 
static size_t image_curl_read_callback(void *dest, size_t size, size_t nmemb, void *userp)
{
  struct WriteThis *wt = (struct WriteThis *)userp;
  size_t buffer_size = size*nmemb;
 
  if(wt->sizeleft) {
    /* copy as much as possible from the source to the destination */ 
    size_t copy_this_much = wt->sizeleft;
    if(copy_this_much > buffer_size)
      copy_this_much = buffer_size;
    memcpy(dest, wt->readptr, copy_this_much);
 
    wt->readptr += copy_this_much;
    wt->sizeleft -= copy_this_much;
    return copy_this_much; /* we copied this many bytes */ 
  }
 
  return 0; /* no more data left to deliver */ 
}

struct CurlThreadInfo {
	char url[1024];
	void *data;
	pthread_t thread_id;
	size_t thread_num;
	size_t data_size;
};

void * SendDataToURL(void *args) {
	struct CurlThreadInfo *curl_info = (struct CurlThreadInfo *) args;
 	/* get a curl handle */ 
	CURL *curl = curl_easy_init();
  CURLcode res;
	if(curl) {
		/* First set the URL that is about to receive our POST. This URL can
			just as well be a https:// URL if that is what should receive the
			data. */ 
 
    /* set our custom set of headers */ 
		struct curl_slist *curl_header = NULL;
		curl_header = curl_slist_append(curl_header, "Content-Type: image/png");
		curl_easy_setopt(curl, CURLOPT_HTTPHEADER, curl_header);
		
		curl_easy_setopt(curl, CURLOPT_URL, curl_info->url);
		/* Now specify the PUT data */ 
		curl_easy_setopt(curl, CURLOPT_PUT, 1L);
		
		curl_easy_setopt(curl, CURLOPT_READFUNCTION, image_curl_read_callback);

		curl_easy_setopt(curl, CURLOPT_INFILESIZE, curl_info->data_size);
	
		struct WriteThis wt;
 
		wt.readptr = (char *)curl_info->data;
		wt.sizeleft = curl_info->data_size;
		curl_easy_setopt(curl, CURLOPT_READDATA, &wt);

		/* Perform the request, res will get the return code */ 
		res = curl_easy_perform(curl);
		if(res != CURLE_OK) {
      fprintf(stderr, "curl_easy_perform() failed: %s\n",
							curl_easy_strerror(res));
			return (void *)1;
		}
 
		curl_slist_free_all(curl_header);
    /* always cleanup */ 
    curl_easy_cleanup(curl);
	}
	else {
		cerr << "Curl Failed to Initialize" << endl;
		return (void *)1;
	}
	return NULL;
}

int CheckForQueue(Redis &redis, char *queue_name, char *working_name,
		char *queue_details_key, int32_t *queue_id, int32_t *start_lat,
		int32_t *start_lng, double *multiply_const, int32_t *image_size,
		double *quality_scale, enum QualityCalcMethod *quality_calc_method, 
		double *quality_calc_value, char *polygons_db_request, char *aws_s3_url) {
	try {
    auto id = redis.brpoplpush(queue_name, working_name, 30);
		if(id) {
			unordered_map<string, string> m;
			sscanf((*id).c_str(), "%d", queue_id);
			sprintf(queue_details_key, "%s:%d", REDIS_QUEUE_DETAILS_BASE_NAME, *queue_id);
			auto queue_details = redis.get(queue_details_key);
			int temp_for_enum;
			if(!queue_details) {
				redis.lrem(working_name, 1, (*id).c_str());
				return 0;
			}
			sscanf((*queue_details).c_str(), "%d %d %lf %d %lf %d %lf %[^\n] %[^\n]",
				start_lat, 
				start_lng,
				multiply_const,
				image_size,
				quality_scale,
				&temp_for_enum,
				quality_calc_value,
				aws_s3_url,
				polygons_db_request
			);
			*quality_calc_method = (enum QualityCalcMethod) temp_for_enum;
			return 1;
		}
		else
			return 0;
	} catch (const Error &e) {
		cerr << e.what() << endl;
		return -1;
	}
}

int has_sig_inted = 0;

static void hdl (int sig, siginfo_t *siginfo, void *context)
{
	has_sig_inted = 1;
}

/**
* Program main
*/
int main(int argc, char **argv) {
	struct sigaction act;
 
	memset (&act, '\0', sizeof(act));
 
	/* Use the sa_sigaction field because the handles has two additional parameters */
	act.sa_sigaction = &hdl;
 
	/* The SA_SIGINFO flag tells sigaction() to use the sa_sigaction field, not sa_handler. */
	act.sa_flags = SA_SIGINFO;

	if (sigaction(SIGINT, &act, NULL) < 0) {
		perror ("sigaction");
		return 1;
	}

	if (sigaction(SIGTERM, &act, NULL) < 0) {
		perror ("sigaction");
		return 1;
	}
 
	const char *REDIS_URL, *PG_URL;
	// Check ENV Vars
	if(!(REDIS_URL = getenv("REDIS_URL")) || !REDIS_URL[0]) {
		cerr << "Missing REDIS_URL" << endl;
		fprintf(stderr, "Missing REDIS_URL\n");
		exit(1);
	}
	if(!(PG_URL = getenv("PG_URL")) || !PG_URL[0]) {
		cerr << "Missing PG_URL" << endl;
		exit(1);
	}

	// Parse Redis URL
	char REDIS_PASSWORD[128] = "", REDIS_HOST[128], REDIS_PORT[8] = "";
	const char *loc_of_at, *loc_of_colon;
	if(loc_of_at = strchr(REDIS_URL,'@')) {
		loc_of_colon = strchr(REDIS_URL+8,':'); // : after "redis://"
		if(loc_of_colon && loc_of_colon < loc_of_at) {
			sscanf(REDIS_URL,"redis://%*[^:]:%[^@]@%[^:]:%s", REDIS_PASSWORD, REDIS_HOST, REDIS_PORT);
		}
		else {
			sscanf(REDIS_URL,"redis://%[^@]@%[^:]:%s", REDIS_PASSWORD, REDIS_HOST, REDIS_PORT);
		}
	}
	else {
		sscanf(REDIS_URL,"redis://%[^:]:%s", REDIS_HOST, REDIS_PORT);
	}
	int32_t *vectors = NULL;
	double *poly_values = NULL;
	int32_t num_polys, start_lat, start_lng, image_size, total_length;
	enum QualityCalcMethod quality_calc_method;
	double quality_scale, quality_calc_value, multiply_const;
	int32_t *vector_lengths = NULL;
	int32_t queue_id;
	char polygons_db_request[MAX_POSTGRES_QUERY_SIZE];
	char complete_key[64];
	char queue_details_key[64];
	char id_str[32];
	int thread_status;
	size_t i;

	/* In windows, this will init the winsock stuff */ 
	curl_global_init(CURL_GLOBAL_ALL);
	
	// DB connections setup
	connection *postgres_connection;
	ConnectionOptions connection_options; // Redis connection
	connection_options.host = REDIS_HOST;
	if(REDIS_PORT[0])
		connection_options.port = atoi(REDIS_PORT);
	if(REDIS_PASSWORD[0])
		connection_options.password = REDIS_PASSWORD;
	connection_options.keep_alive = true;
	Redis *redis;

	try {
		redis = new Redis(connection_options);
		postgres_connection = new connection(PG_URL);
		if (postgres_connection->is_open()) {
			cout << "Opened PG database successfully: " << postgres_connection->dbname() << endl;
		} else {
			cerr << "Can't open database" << endl;
			exit(1);
		}
	} catch (const Error &e) {
		cerr << e.what() << endl;
		exit(1);
	}
	auto pipe = redis->pipeline();
	string queue_data;
	int queue_status;
	struct CurlThreadInfo curl_threads[PNG_POOL_SIZE] = {0};
	for(i = 1; i <= PNG_POOL_SIZE; i++)
		curl_threads[i].thread_num = i;

	void *thread_res;
	int current_png = 0;
	while(1)
	{
		if(has_sig_inted) {
			cout << "Received SIGINT or TERM and gracefully quiting" << endl;
			break;
		}
		// cout << "Waiting for Queue\n";
		queue_status = CheckForQueue(
			*redis, 
			(char *) REDIS_QUEUE_NAME,
			(char *) REDIS_WORKING_NAME,
			queue_details_key,
			&queue_id,
			&start_lat,
			&start_lng,
			&multiply_const,
			&image_size,
			&quality_scale,
			&quality_calc_method,
			&quality_calc_value,
			polygons_db_request,
			curl_threads[current_png].url
		);
		if(queue_status == 0) { // Redis Blocking Timeout
			cout << "Clearing CURL Queue" << endl;

			for (i = 0; i < current_png; i++) {
				if(thread_status = pthread_join(curl_threads[i].thread_id, &thread_res)) {
					cerr << "Error joining thread: " << thread_status << endl;
					exit(1);
				}
				free(curl_threads[i].data);
				curl_threads[i].data = NULL;
			}
			current_png = 0;
			continue;
		}
		else if(queue_status == -1) {
			exit(1);
		}

		RetrieveValuesFromPG(
			postgres_connection,
			polygons_db_request,
			multiply_const,
			&vectors,
			&total_length,
			&poly_values,
			&num_polys,
			&vector_lengths
		);

		cout << "Calculating...\n";

		if(PointInPolygonsImage(&curl_threads[current_png].data, &curl_threads[current_png].data_size, start_lat, start_lng, image_size, quality_scale, quality_calc_method, quality_calc_value,
				vectors, total_length, poly_values, num_polys, vector_lengths))
			exit(1);
		if(vectors) {
			free(vectors);
			vectors = NULL;
		}
		if(poly_values) {
			free(poly_values);
			poly_values = NULL;
		}
		if(vector_lengths) {
			free(vector_lengths);
			vector_lengths = NULL;
		}
		if(curl_threads[current_png].data) {
			printf("Sending png image of size: %.2f KB\n", curl_threads[current_png].data_size/1024.0);
			if (thread_status = pthread_create(&curl_threads[current_png].thread_id, NULL,
				&SendDataToURL, &curl_threads[current_png])) {
				cerr << "Error creating thread: " << thread_status << endl;
				exit(1);
			}
			if(++current_png >= PNG_POOL_SIZE) {
				current_png = 0;
				cout << "Clearing CURL Queue" << endl;

				for (i = 0; i < PNG_POOL_SIZE; i++) {
					if(thread_status = pthread_join(curl_threads[i].thread_id, &thread_res)) {
						cerr << "Error joining thread: " << thread_status << endl;
						exit(1);
					}
					free(curl_threads[i].data);
					curl_threads[i].data = NULL;
				}
			}
		}
		else {
			cout << "No image to send\n";
		}
		sprintf(complete_key, "%s:%d", REDIS_COMPLETE_BASE_NAME, queue_id);
		sprintf(id_str, "%d", queue_id);
		try {
			pipe.lpush(complete_key, "success").
			expire(complete_key, 300).
			lrem(REDIS_WORKING_NAME, 1, id_str).
			del(queue_details_key).
			exec();
		} catch (const Error &e) {
			cerr << e.what() << endl;
			exit(1);
		}
		
	}
	cout << "Clearing CURL Queue" << endl;
	for (i = 0; i < current_png; i++) {
		if(thread_status = pthread_join(curl_threads[i].thread_id, &thread_res)) {
			cerr << "Error joining thread: " << thread_status << endl;
			exit(1);
		}
		free(curl_threads[i].data);
	}
	try {
		postgres_connection->disconnect();
	} catch (const Error &e) {
		cerr << e.what() << endl;
		exit(1);
	}
	delete postgres_connection;
	delete redis;
	curl_global_cleanup();
	exit(0);
}